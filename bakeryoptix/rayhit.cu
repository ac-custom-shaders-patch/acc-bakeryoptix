#include "hip/hip_runtime.h"
/* 
 * Copyright (c) 2013-2018, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

#include "ray.h"

#include <optix.h>
#include <optixu/optixu_math_namespace.h>

// Context global variables provided by the renderer system.
rtDeclareVariable(rtObject, sysTopObject, , );

// Semantic variables.
rtDeclareVariable(optix::Ray, theRay, rtCurrentRay, );

rtDeclareVariable(PerRayData, thePrd, rtPayload, );

// Attributes.
rtDeclareVariable(optix::float3, varGeoNormal, attribute GEO_NORMAL, );
//rtDeclareVariable(optix::float3, varTangent,   attribute TANGENT, );
rtDeclareVariable(optix::float3, varNormal,    attribute NORMAL, );
//rtDeclareVariable(optix::float3, varTexCoord,  attribute TEXCOORD, ); 

// This closest hit program only uses the geometric normal and the shading normal attributes.
// OptiX will remove all code from the intersection programs for unused attributes automatically.

// Note that the matching between attribute outputs from the intersection program and 
// the inputs in the closesthit and anyhit programs is done with the type (here float3) and
// the user defined attribute semantic (e.g. here NORMAL). 
// The actual variable name doesn't need to match but it's recommended for clarity.

RT_PROGRAM void rayhit()
{
  // Transform the (unnormalized) object space normals into world space.
  float3 geoNormal = optix::normalize(rtTransformNormal(RT_OBJECT_TO_WORLD, varGeoNormal));
  float3 normal    = optix::normalize(rtTransformNormal(RT_OBJECT_TO_WORLD, varNormal));

  // Check if the ray hit the geometry on the frontface or the backface.
  // The geometric normal is always defined on the front face of the geometry.
  // In this implementation the coordinate systems are right-handed and the frontface triangle winding is counter-clockwise (matching OpenGL).

  // If theRay.direction and geometric normal are in the same hemisphere we're looking at a backface.
  if (0.0f < optix::dot(theRay.direction, geoNormal))
  {
    // Flip the shading normal to the backface, because only that is used below.
    // (See later examples for more intricate handling of the frontface condition.)
    normal = -normal;
  }

  // Visualize the resulting world space normal on the surface we're looking on.
  // Transform the normal components from [-1.0f, 1.0f] to the range [0.0f, 1.0f] to get colors for negative values.
  // thePrd.radiance = normal * 0.5f + 0.5f;
  thePrd.radiance = make_float3(0.f, 0.f, 0.f);
}