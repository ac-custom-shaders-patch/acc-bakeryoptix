#include "hip/hip_runtime.h"
/* 
 * Copyright (c) 2013-2018, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

#include "ray.h"

#include <optix.h>
#include <optixu/optixu_math_namespace.h>

// Context global variables provided by the renderer system.
rtDeclareVariable(rtObject, sysTopObject, , );

// Semantic variables.
rtDeclareVariable(optix::Ray, theRay, rtCurrentRay, );
rtDeclareVariable(float, theIntersectionDistance, rtIntersectionDistance, );
rtDeclareVariable(PerRayData, thePrd, rtPayload, );

// Attributes.
rtDeclareVariable(optix::float3, varGeoNormal, attribute GEO_NORMAL, );
//rtDeclareVariable(optix::float3, varTangent,   attribute TANGENT, );
rtDeclareVariable(optix::float3, varNormal, attribute NORMAL, );
rtDeclareVariable(optix::float3, varTexCoord, attribute TEXCOORD, );

// Material parameter definition.
rtDeclareVariable(float, parMaterialAlbedo, , ); // Per Material index into the sysMaterialParameters array.
rtDeclareVariable(int, parMaterialTexture, , );  // Per Material index into the sysMaterialParameters array.
// 
// This closest hit program only uses the geometric normal and the shading normal attributes.
// OptiX will remove all code from the intersection programs for unused attributes automatically.

// Note that the matching between attribute outputs from the intersection program and 
// the inputs in the closesthit and anyhit programs is done with the type (here float3) and
// the user defined attribute semantic (e.g. here NORMAL). 
// The actual variable name doesn't need to match but it's recommended for clarity.

RT_FUNCTION void alignVector(float3 const& axis, float3& w)
{
	// Align w with axis.
	const float s = copysign(1.0f, axis.z);
	w.z *= s;
	const float3 h = make_float3(axis.x, axis.y, axis.z + s);
	const float k = optix::dot(w, h) / (1.0f + fabsf(axis.z));
	w = k * h - w;
}

RT_FUNCTION void unitSquareToCosineHemisphere(const float2 sample, float3 const& axis, float3& w, float& pdf)
{
	// Choose a point on the local hemisphere coordinates about +z.
	const float theta = 2.0f * M_PIf * sample.x;
	const float r = sqrtf(sample.y);
	w.x = r * cosf(theta);
	w.y = r * sinf(theta);
	w.z = 1.0f - w.x * w.x - w.y * w.y;
	w.z = (0.0f < w.z) ? sqrtf(w.z) : 0.0f;

	pdf = w.z * M_1_PIf;

	// Align with axis.
	alignVector(axis, w);
}

RT_PROGRAM void rayhit()
{
	// Transform the (unnormalized) object space normals into world space.
	float3 geoNormal = optix::normalize(rtTransformNormal(RT_OBJECT_TO_WORLD, varGeoNormal));
	float3 normal = optix::normalize(rtTransformNormal(RT_OBJECT_TO_WORLD, varNormal));

	thePrd.pos = theRay.origin + theRay.direction * theIntersectionDistance; // Advance the path to the hit position in world coordinates.

	// Explicitly include edge-on cases as frontface condition! (Important for nested materials shown in a later example.)
	thePrd.flags |= (0.0f <= optix::dot(thePrd.wo, geoNormal)) ? FLAG_FRONTFACE : 0;

	if ((thePrd.flags & FLAG_FRONTFACE) == 0) // Looking at the backface?
	{
		// Means geometric normal and shading normal are always defined on the side currently looked at.
		// This gives the backfaces of opaque BSDFs a defined result.
		geoNormal = -geoNormal;
		normal = -normal;
		// Do not recalculate the frontface condition!
	}

	// A material system with support for arbitrary mesh lights would evaluate its emission here.
	thePrd.radiance = make_float3(0.0f);

	// Start fresh with the next BSDF sample.  (Either of these values remaining zero is an end-of-path condition.)
	thePrd.f_over_pdf = make_float3(0.0f);
	thePrd.pdf = 0.0f;

	// Lambert sampling: Cosine weighted hemisphere sampling above the shading normal.
	// This calculates the ray.direction for the next path segment in wi and its probability density function value in pdf.
	unitSquareToCosineHemisphere(rng2(thePrd.seed), normal, thePrd.wi, thePrd.pdf);

	// Do not sample opaque surfaces below the geometry!
	// Mind that the geometry normal has been flipped to the side the ray points at.
	if (thePrd.pdf <= 0.0f || optix::dot(thePrd.wi, geoNormal) <= 0.0f)
	{
		thePrd.flags |= FLAG_TERMINATE;
		return;
	}

	// MaterialParameter parameters = sysMaterialParameters[parMaterialIndex];

	// This would be the universal implementation for an arbitrary sampling of a diffuse surface.
	// thePrd.f_over_pdf = parameters.albedo * (M_1_PIf * fabsf(optix::dot(prd.wi, normal)) / prd.pdf); 

	// PERF Since the cosine-weighted hemisphere distribution is a perfect importance-sampling of the Lambert material,
	// the whole term ((M_1_PIf * fabsf(optix::dot(prd.wi, normal)) / prd.pdf) is always 1.0f here!
	thePrd.f_over_pdf = make_float3(parMaterialAlbedo, parMaterialAlbedo, parMaterialAlbedo);

	// This is a brute-force path tracer. There is no next event estimation (direct lighting) here.
	// Note that because of that, the albedo affects the path throughput only.
	// This material is not returning any radiance because it's not a light source.
}
