#include "hip/hip_runtime.h"
/* 
 * Copyright (c) 2013-2018, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

#include "ray.h"
#include <optixu/optixu_aabb_namespace.h>
#include <optixu/optixu_math_namespace.h>

rtBuffer<float3> attributesBuffer;
rtBuffer<uint3> indicesBuffer;

// Axis Aligned Bounding Box routine for indexed interleaved triangle data.
RT_PROGRAM void raybb(int primitiveIndex, float result[6])
{
  const uint3 indices = indicesBuffer[primitiveIndex];

  const float3 v0 = attributesBuffer[indices.x];
  const float3 v1 = attributesBuffer[indices.y];
  const float3 v2 = attributesBuffer[indices.z];
  const float area = optix::length(optix::cross(v1 - v0, v2 - v0));

  optix::Aabb *aabb = (optix::Aabb *) result;

  if (0.0f < area && !isinf(area))
  {
    aabb->m_min = fminf(fminf(v0, v1), v2);
    aabb->m_max = fmaxf(fmaxf(v0, v1), v2);
  }
  else
  {
    aabb->invalidate();
  }
}