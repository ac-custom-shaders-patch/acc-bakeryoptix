#include "hip/hip_runtime.h"
/* 
 * Copyright (c) 2013-2018, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

#include <builtin_types.h>

#include "ray.h"

#include <optix.h>
#include <optixu/optixu_math_namespace.h>

struct vertex
{
	float3 pos;
	float3 tex;
};

rtBuffer<vertex> attributesBuffer;
rtBuffer<uint3> indicesBuffer;

// Attributes.
rtDeclareVariable(optix::float3, varGeoNormal, attribute GEO_NORMAL, );
//rtDeclareVariable(optix::float3, varTangent, attribute TANGENT, );
rtDeclareVariable(optix::float3, varNormal, attribute NORMAL, );
rtDeclareVariable(optix::float3, varTexCoord, attribute TEXCOORD, );

rtDeclareVariable(optix::Ray, theRay, rtCurrentRay, );

// Intersection routine for indexed interleaved triangle data.
RT_PROGRAM void rayintersection(int primitiveIndex)
{
	const uint3 indices = indicesBuffer[primitiveIndex];

	vertex const& a0 = attributesBuffer[indices.x];
	vertex const& a1 = attributesBuffer[indices.y];
	vertex const& a2 = attributesBuffer[indices.z];

	const vertex v0 = a0;
	const vertex v1 = a1;
	const vertex v2 = a2;

	float3 n;
	float t;
	float beta;
	float gamma;

	if (intersect_triangle(theRay, v0.pos, v1.pos, v2.pos, n, t, beta, gamma))
	{
		if (rtPotentialIntersection(t))
		{
			// Barycentric interpolation:
			const float alpha = 1.0f - beta - gamma;

			// Note: No normalization on the TBN attributes here for performance reasons.
			//       It's done after the transformation into world space anyway.
			varGeoNormal = n;
			//varTangent        = a0.tangent  * alpha + a1.tangent  * beta + a2.tangent  * gamma;
			//varNormal         = a0.normal   * alpha + a1.normal   * beta + a2.normal   * gamma;
			varNormal = n;
			varTexCoord = a0.tex * alpha + a1.tex * beta + a2.tex * gamma;

			rtReportIntersection(0);
		}
	}
}
